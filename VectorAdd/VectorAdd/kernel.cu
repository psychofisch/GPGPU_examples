#include <hip/hip_runtime.h>

__global__ void cudaVectorAddKernel(const int *vectorA, const int *vectorB, int *vectorC, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		vectorC[i] = vectorA[i] + vectorB[i];
	}
}

extern "C" void cudaVectorAdd(const int *vectorA, const int *vectorB, int *vectorC, int numElements)
{
	int *cudaA, *cudaB, *cudaC;
	hipMalloc((void**)&cudaA, sizeof(int) *  numElements);
	hipMalloc((void**)&cudaB, sizeof(int) *  numElements);
	hipMalloc((void**)&cudaC, sizeof(int) *  numElements);

	hipMemcpy(cudaA, vectorA, sizeof(int) *  numElements, hipMemcpyDefault);
	hipMemcpy(cudaB, vectorB, sizeof(int) * numElements, hipMemcpyDefault);

	int threads = 256;
	int blocks = (numElements + threads - 1) / threads;

	cudaVectorAddKernel <<< threads, blocks >>> (cudaA, cudaB, cudaC, numElements);

	hipMemcpy(vectorC, cudaC, sizeof(int) *  numElements, hipMemcpyDefault);

	hipFree((void**)&cudaA);
	hipFree((void**)&cudaB);
	hipFree((void**)&cudaC);
}
