#include "hip/hip_runtime.h"
#include "ThrustHelper.h"

inline __host__ __device__ bool operator==(float3& lhs, float3& rhs)
{
	if (lhs.x == rhs.x && lhs.y == rhs.y && lhs.z == rhs.z)
		return true;
	else
		return false;
}

ThrustHelper::SimulationFunctor::SimulationFunctor(float dt_, float3 dim_, float3 g_, SimulationData simData_)
	:dt(dt_),
	dimension(dim_),
	gravity(g_),
	simData(simData_)
{}

__host__ __device__ float4 ThrustHelper::SimulationFunctor::operator()(float4 outerPos, float4 outerVel)
{
	float3 particlePosition = make_float3(outerPos);
	float3 particleVelocity = make_float3(outerVel);

	float3 result = make_float3(0.f);

	particleVelocity += gravity * dt;

	// static collision
	//TODO: write some kind of for-loop
	if ((particlePosition.x + particleVelocity.x * dt > dimension.x && particleVelocity.x > 0.f) || (particlePosition.x + particleVelocity.x * dt < 0.f && particleVelocity.x < 0.f))
	{
		if (particlePosition.x + particleVelocity.x * dt < 0.f)
			particlePosition.x = 0.f;
		else
			particlePosition.x = dimension.x;

		particleVelocity.x *= -.3f;
	}

	if ((particlePosition.y + particleVelocity.y * dt  > dimension.y && particleVelocity.y > 0.f) || (particlePosition.y + particleVelocity.y * dt < 0.f && particleVelocity.y < 0.f))
	{
		if (particlePosition.y + particleVelocity.y * dt < 0.f)
			particlePosition.y = 0.f;
		else
			particlePosition.y = dimension.y;

		particleVelocity.y *= -.3f;
	}

	if ((particlePosition.z + particleVelocity.z * dt > dimension.z && particleVelocity.z > 0.f) || (particlePosition.z + particleVelocity.z * dt < 0.f && particleVelocity.z < 0.f))
	{
		if (particlePosition.z + particleVelocity.z * dt < 0.f)
			particlePosition.z = 0.f;
		else
			particlePosition.z = dimension.z;

		particleVelocity.z *= -.3f;
	}
	// *** sc

	// particleVelocity += dt * particleVelocity * -0.01f;//damping
	particlePosition += particleVelocity * dt;

	//positionOut[index] = make_float4(particlePosition);

	return make_float4(particlePosition);
}

ThrustHelper::PressureFunctor::PressureFunctor(float3 pos_, float3 vel_, SimulationData simData_)
	:pos(pos_),
	vel(vel_),
	simData(simData_)
{}

__host__ __device__ float4 ThrustHelper::PressureFunctor::operator()(float4 outerPos, float4 outerVel)
{
	float3 pressureVec = make_float3(0.f);
	float3 viscosityVec = pressureVec;
	float influence = 0.f;

	float3 dirVec = pos - make_float3(outerPos);
	float dist = length(dirVec);//TODO: maybe use half_length

	if (dist > simData.interactionRadius || dist == 0.0f)
		return make_float4(0.f);

	float3 dirVecN = normalize(dirVec);
	float moveDir = dot(vel - make_float3(outerVel), dirVecN);
	float distRel = 1.0f - dist / simData.interactionRadius;

	float sqx = distRel * distRel;

	influence += 1.0f;

	// viscosity
	if (true || moveDir > 0)
	{
		float factor = sqx * (simData.viscosity * moveDir);
		float3 impulse = factor * dirVecN;
		viscosityVec -= impulse;
	}
	// *** v

	float pressure = sqx * simData.pressureMultiplier;

	pressureVec += (pressure - simData.restPressure) * dirVecN;

	//compress viscosity TODO: fix the root of this problem and not just limit it manually
	//float threshold = 50.0;
	if (influence > 0.f)
	{
		viscosityVec = viscosityVec / influence;
	}

	if (length(viscosityVec) > 100.0)
		viscosityVec = normalize(viscosityVec) * 100.0;
	//*** lv

	return make_float4(pressureVec + viscosityVec);
}

std::unique_ptr<ThrustHelper::ThrustParticleData> ThrustHelper::setup(uint numberOfParticles)
{
	auto r = std::make_unique<ThrustHelper::ThrustParticleData>();
	r->position.reserve(numberOfParticles);
	r->velocity.reserve(numberOfParticles);
	r->positionOut.reserve(numberOfParticles);
	return r;
}

void ThrustHelper::thrustParticleUpdate(
	ThrustParticleData& tdata,
	float4* position,
	float4* positionOut,
	float4* velocity,
	const float dt,
	const float3 gravity,
	const float3 dimension,
	const uint numberOfParticles,
	SimulationData simData)
{
	/*thrust::device_vector<float4> devicePos(position, position + numberOfParticles);
	thrust::device_vector<float4> deviceVel(velocity, velocity + numberOfParticles);
	thrust::device_vector<float4> deviceOut(numberOfParticles);*/
	tdata.position.assign(position, position + numberOfParticles);
	tdata.velocity.assign(velocity, velocity + numberOfParticles);
	tdata.positionOut.resize(numberOfParticles);

	// because "nested" thrust-transforms calls are not allowed and any other method would use N� memory I decided to step through the particle array via a CPU for-loop
	for (uint i = 0; i < numberOfParticles; ++i)
	{
		float3 particlePosition = make_float3(position[i]);
		float3 particleVelocity = make_float3(velocity[i]);

		// calculate pressure
		thrust::transform(tdata.position.begin(), tdata.position.end(), tdata.velocity.begin(), tdata.positionOut.begin(), PressureFunctor(particlePosition, particleVelocity, simData));
		//float4 pressure4 = make_float4(0.0);
		float4 pressure4 = thrust::reduce(tdata.positionOut.begin(), tdata.positionOut.end(), make_float4(0.0f), thrust::plus<float4>());//sums all values together

		particleVelocity += (gravity + make_float3(pressure4)) * dt;

		// static collision
		//TODO: write some kind of for-loop
		if ((particlePosition.x + particleVelocity.x * dt > dimension.x && particleVelocity.x > 0.f) || (particlePosition.x + particleVelocity.x * dt < 0.f && particleVelocity.x < 0.f))
		{
			if (particlePosition.x + particleVelocity.x * dt < 0.f)
				particlePosition.x = 0.f;
			else
				particlePosition.x = dimension.x;

			particleVelocity.x *= -.3f;
		}

		if ((particlePosition.y + particleVelocity.y * dt > dimension.y && particleVelocity.y > 0.f) || (particlePosition.y + particleVelocity.y * dt < 0.f && particleVelocity.y < 0.f))
		{
			if (particlePosition.y + particleVelocity.y * dt < 0.f)
				particlePosition.y = 0.f;
			else
				particlePosition.y = dimension.y;

			particleVelocity.y *= -.3f;
		}

		if ((particlePosition.z + particleVelocity.z * dt > dimension.z && particleVelocity.z > 0.f) || (particlePosition.z + particleVelocity.z * dt < 0.f && particleVelocity.z < 0.f))
		{
			if (particlePosition.z + particleVelocity.z * dt < 0.f)
				particlePosition.z = 0.f;
			else
				particlePosition.z = dimension.z;

			particleVelocity.z *= -.3f;
		}
		// *** sc

		positionOut[i] = make_float4(particlePosition + particleVelocity * dt);
		velocity[i] = make_float4(particleVelocity);
	}
	// calculate simulation
	//thrust::transform(devicePos.begin(), devicePos.end(), deviceVel.begin(), deviceOut.begin(), SimulationFunctor(dt, dimension, gravity, simData));
	
	//thrust::copy(tdata.positionOut.begin(), tdata.positionOut.end(), positionOut);
}
