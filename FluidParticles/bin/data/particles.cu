#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
 
typedef unsigned int uint;

float3 calculatePressure(float3* positions, uint1 index, uint numberOfParticles, float smoothingWidth);

__global__ void particleUpdate(
	float3* positions, 
	float3* velocity, 
	const float dt, 
	const float smoothingWidth, 
	const float3 gravity,
	const float4 dimension,
	const uint numberOfParticles)
{

}

float3 calculatePressure(float3* positions, uint index, uint numberOfParticles, float smoothingWidth)
{
	float3 particlePosition = positions[index];

	float3 pressureVec = make_float3(0,0,0);
	for (uint i = 0; i < numberOfParticles; i++)
	{
		if (index == i)
			continue;

		//float3 dirVec = particlePosition - positions[i];
		//float dist = length(dirVec);//TODO: maybe use half_length

		//if (dist > smoothingWidth * 1.0f)
		//	continue;

		//float pressure = 1.f - (dist / smoothingWidth);
		////float pressure = amplitude * exp(-dist / smoothingWidth);

		//pressureVec += (float4)(pressure * normalize(dirVec), 0.f);
		//// pressureVec += vec4(dirVec, 0.f);

		//pressureVec.w = dist;

		//break;
	}

	return pressureVec;
}
